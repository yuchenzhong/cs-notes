
#include <hip/hip_runtime.h>
__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) 
        y[i] += x[i];
}

int main(void) {
    int N = 1 << 20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    for (int i = 0; i < 3; ++i) {
        add<<<numBlocks, blockSize>>>(N, x, y);
    }

    hipDeviceSynchronize();
}
